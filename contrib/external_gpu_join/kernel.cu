#include "kernel.cuh"
#include "stdio.h"
#include "assert.h"

#define unsigned long ul
#define unsigned int ui
hipError_t cudaStatus[10];
hipDeviceProp_t GPUprop;
ul SupportedKBlocks, SupportedMBlocks, MaxTherPerBlk;
char SupportedBlocks[100];

/*
* TODO
* add 全局数组指针
* 申请内存
* 数据拷贝至显存
* join
* 数据拷回内存

*/

void check()
{
    int NumGPUs = 0;
    hipGetDeviceCount(&NumGPUs);
    if (!NumGPUs) {
        printf("\n No GPU Device is available\n");
        exit(EXIT_FAILURE);  // exit
    }
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        exit(EXIT_FAILURE);  // exit
    }
    hipGetDeviceProperties(&GPUprop, 0);
    SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
    SupportedMBlocks = SupportedKBlocks / 1024;

    sprintf(SupportedBlocks,
        "%u %c",
        (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks,
        (SupportedMBlocks >= 5) ? 'M' : 'K');
}

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity - 1);
}

std::size_t* p_size[2];
Datum* GpuColBuffer[10];

void moveTupletoGPU(void* arg)
{
    ExternalJoinState* ejs = static_cast<ExternalJoinState*>(arg);

    /* if TupleBufferQueue is finalized, TupleBufferQueue->getLength() returns -1 */
    int cnt = 0;
    while (ejs->tbq.getLength() >= 0) {

        ColBuffer* tb = ejs->tbq.pop();
        std::size_t size;

        /* wait for scan completion */
        //        pthread_testcancel();
        if (tb == NULL) {
            ::usleep(1);
            continue;
        }
        size = tb->getContentSize();
        /* send tuple buffer size to external */
        // sendStrong(sock, &size, sizeof(size));

        int ncol = tb->totalAttr;
        for (int i = 0; i < ncol; i++) {
            // for (int j = 0; j < col[i].size(); j++) {
            //     /*copy data to device*/
            // }
            
            /*暂时忽略类型， Datum 貌似可用， 参见heaptuple.cpp*/
            cudaStatus[i] = hipMalloc(&GpuColBuffer[i], col[i].size());
            if (cudaStatus[i]) {
            }
            cuda
        }

        /*build hash table here?

        然后在Exec：   join 的时候 一条一条探测。
        */
        cudaStatus = hipMalloc((void**)&p_size[cnt], sizeof(std::size_t));

        /* send tuples to external */
        //    sendStrong(sock, tb->getBufferPointer(), size);

        cudaStatus2 = hipMalloc((void**)&GpuBuffer[cnt], sizeof(size));
        TupleBuffer::destructor(tb);
        cnt++;
    }
    return NULL;
}

void moveTupletoHost()
{}
