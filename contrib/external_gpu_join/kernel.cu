#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "stdio.h"
#include "assert.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

hipDeviceProp_t GPUprop;
/*
* TODO
* add 全局数组指针
* 申请内存
* 数据拷贝至显存
* join
* 数据拷回内存

*/

// void check()
// {
//     int NumGPUs = 0;
//     hipGetDeviceCount(&NumGPUs);
//     if (!NumGPUs) {
//         printf("\n No GPU Device is available\n");
//         exit(EXIT_FAILURE);  // exit
//     }
//     hipError_t cudaStatus = hipSetDevice(0);
//     if (cudaStatus != hipSuccess) {
//         exit(EXIT_FAILURE);  // exit
//     }
//     hipGetDeviceProperties(&GPUprop, 0);
// }

// 32 bit Murmur3 hash
// __device__ uint32_t hash(uint32_t k)
// {
//     k ^= k >> 16;
//     k *= 0x85ebca6b;
//     k ^= k >> 13;
//     k *= 0xc2b2ae35;
//     k ^= k >> 16;
//     return k & (kHashTableCapacity - 1);
// }


// bool tupleKvcudaMallocu(Tuplekv * p, size_t t){
//     hipError_t cudaStatus = hipMalloc((void**)&p, t);
//     if (cudaStatus != hipSuccess) {
//         /*call error func*/
//         return true;
//     }
//     return false;
// }
// bool cudaMemcpytoDevice(){

// }
// bool cudaMemcpytoHost(){

// }

__global__ void nLJ(struct Tuplekv* d_a, struct Tuplekv* d_b, long n_a, long n_b, struct Result* res)
{
    // int x = threadIdx.x + blockIdx.x * blockDim.x;
    // int y = threadIdx.y + blockIdx.y * blockDim.y;
    // if (x < n_a && y < n_b) {
    //     if (d_a[x].key == d_b[y].key) {
    //         (*res)->key1 = d_b[x].key;
    //         res->dval1 = d_a[x].dval;
    //         res->key2 = d_a[y].key;
    //         res->dval2 = d_a[y].dval;
    //     } else
    //         res = NULL;
    // }
    long x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x >= n_b)
        return;
    for (long i = 0; i < n_a; i++) {
        if (d_b[x].key == d_a[i].key) {
            (res + i + n_a * x)->key1 = d_a[i].key;
            (res + i + n_a * x)->dval1 = d_a[i].dval;
            (res + i + n_a * x)->key2 = d_b[x].key;
            (res + i + n_a * x)->dval2 = d_b[x].dval;
        } else {
            (res + i + n_a * x)->key1 = -1;
            (res + i + n_a * x)->dval1 = -1;
            (res + i + n_a * x)->key2 = -1;
            (res + i + n_a * x)->dval2 = -1;
        }
    }
}



void nestLoopJoincu(struct Tuplekv* d_a, struct Tuplekv* d_b, long n_a, long n_b, struct Result* res)
{
    
    nLJ<<<32, 1024>>>(d_a, d_b, n_a, n_b, res);

    hipError_t cudaStatus = hipDeviceSynchronize();

}
