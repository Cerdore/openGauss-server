#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "stdio.h"
#include "assert.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

hipDeviceProp_t GPUprop;


__global__ void nLJ(struct Tuplekv* d_a, struct Tuplekv* d_b, long n_a, long n_b, struct Result* res)
{
    // int x = threadIdx.x + blockIdx.x * blockDim.x;
    // int y = threadIdx.y + blockIdx.y * blockDim.y;
    // if (x < n_a && y < n_b) {
    //     if (d_a[x].key == d_b[y].key) {
    //         (*res)->key1 = d_b[x].key;
    //         res->dval1 = d_a[x].dval;
    //         res->key2 = d_a[y].key;
    //         res->dval2 = d_a[y].dval;
    //     } else
    //         res = NULL;
    // }
    long x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x >= n_b)
        return;
    for (long i = 0; i < n_a; i++) {
        if (d_b[x].key == d_a[i].key) {
            (res + i + n_a * x)->key1 = d_a[i].key;
            (res + i + n_a * x)->dval1 = d_a[i].dval;
            (res + i + n_a * x)->key2 = d_b[x].key;
            (res + i + n_a * x)->dval2 = d_b[x].dval;
        }
    }
}


void nestLoopJoincu(struct Tuplekv* d_a, struct Tuplekv* d_b, long n_a, long n_b, struct Result* res)
{
    nLJ<<<128, 1024>>>(d_a, d_b, n_a, n_b, res);
    hipError_t cudaStatus = hipDeviceSynchronize();
}


/*Hash Join -- simple*/
#define p 334214459
#define TABLESIZE 1000000
#define maxiterations 10
#define KEYEMPTY -1
#define NOTFOUND -100

__device__
struct Tuplekv table[TABLESIZE];



__device__
Tuplekv make_entry(int key, double value){
  //printf("key : %d, value : %d",key , value);
  struct Tuplekv ans = {key, value};
  //printf ("ans : %d ", (int)ans>>32);
  return ans;
}

__device__ int getkey(Tuplekv entry){
  return entry.key;
}

// __device__ unsigned getvalue(unsigned long long entry){
//   return (entry & 0xffffffff) ;
// }

__device__
unsigned hash_function_1(unsigned key){
   int a1 = 5;
   int b1 = 2;
   return (((a1*key+b1)%p)%TABLESIZE);
}

__device__
unsigned hash_function_2(unsigned key){
   int a1 = 13;
   int b1 = 7;
   return (((a1*key+b1)%p)%TABLESIZE);
}

__global__
void hash(struct Tuplekv* d_a, int width, int height){
  
    int index = blockIdx.x * blockDim.x +threadIdx.x;
  //  unsigned long key = Table_A[index*width+0];
  //  unsigned long value = Table_A[index*width+1]; 

    int key = d_a[index].key;
    double value = d_a[index].dval;
    
    Tuplekv entry = make_entry(key,value);
    //printf("entry: %d",entry);
    unsigned location = hash_function_1(key);
    unsigned k = key;
    for (int its = 0; its<maxiterations; its++){
      entry = atomicExch(&table[location], entry);
      
      key = getkey(entry);
      if (key == 0) {
        //printf("key: %lu table: %llu \n",k,table[location]);
        return;
      }
      unsigned location1 = hash_function_1(key);
      unsigned location2 = hash_function_2(key);
      if (location == location1)
      location = location2;
      else if (location == location2)
      location = location1;
    }
    printf("chain was too long");
    return ;
}


// __global__
// void join(int *Table_B,int *Table_C,int width_c,int width,int height){
//   int index = blockIdx.x * blockDim.x +threadIdx.x;
//   unsigned long primkey = Table_B[index*width+0];
//    //printf("primkey : %lu \n",primkey);
//   unsigned long value = Table_B[index*width+1];
//   unsigned location_1 = hash_function_1(primkey);
//   unsigned location_2 = hash_function_2(primkey);
//   unsigned long long entry;
//   if (getkey(entry = table[location_1])!= primkey)
//     if (getkey(entry = table[location_2])!= primkey){
//         entry = make_entry(0,NOTFOUND);
//     }
//  // printf("entry of primkey %lu:%llu \n",primkey,entry);
//   //printf("key from hash table of primkey %lu: %d\n",primkey,getkey(entry));
//   Table_C[index*width_c+0]=getkey(entry);
//   Table_C[index*width_c+1]=getvalue(entry);
//   //printf("key from hash table of primkey %lu: %d\n",primkey,getvalue(entry));
//   Table_C[index*width_c+2] = value;
//   //printf("value from hash table of primkey %lu: %d\n",primkey,value); 
//   for(int l =0 ;l<3 ;l++){
//     //printf("index : %d,Table: %d  ",index,Table_C[index*width_c+l]);
//   }
// }



