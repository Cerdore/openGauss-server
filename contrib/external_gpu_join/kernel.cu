#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "stdio.h"
#include "assert.h"

#define unsigned long ul
#define unsigned int ui
hipError_t cudaStatus[10];
hipDeviceProp_t GPUprop;
ul SupportedKBlocks, SupportedMBlocks, MaxTherPerBlk;
char SupportedBlocks[100];

struct Tuple {
    int key;
    double dval;
};

struct Result {
    int key1;
    double dval1;
    int key2;
    double dval2;
};

/*
* TODO
* add 全局数组指针
* 申请内存
* 数据拷贝至显存
* join
* 数据拷回内存

*/

void check()
{
    int NumGPUs = 0;
    hipGetDeviceCount(&NumGPUs);
    if (!NumGPUs) {
        printf("\n No GPU Device is available\n");
        exit(EXIT_FAILURE);  // exit
    }
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        exit(EXIT_FAILURE);  // exit
    }
    hipGetDeviceProperties(&GPUprop, 0);
    SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
    SupportedMBlocks = SupportedKBlocks / 1024;

    sprintf(SupportedBlocks,
        "%u %c",
        (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks,
        (SupportedMBlocks >= 5) ? 'M' : 'K');
}

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity - 1);
}

void moveTupletoGPU(void* arg)
{
    ExternalJoinState* ejs = static_cast<ExternalJoinState*>(arg);

    /* if TupleBufferQueue is finalized, TupleBufferQueue->getLength() returns -1 */
    int cnt = 0;
    struct* Tuple d_tuple[2];
    while (ejs->tbq.getLength() >= 0) {

        TupleBuffer* tb = ejs->tbq.pop();

        std::size_t size;

        /* wait for scan completion */
        // pthread_testcancel();
        // if (tb == NULL) {
        //     ::usleep(1);
        //     continue;
        // }
        size = tb->getContentSize();

        hipError_t cudaStatus = hipMalloc((void**)&d_tuple[cnt], tb->tupleNum * sizeof(Tuple));
        if (cudaStatus != hipSuccess) {
            /*call error func*/
        }

        cudaStatus = hipMemcpy(d_tuple[cnt], tb->getBufferPointer(), size, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            /*call error func*/
        }

        ejs->d_tuple[cnt] = d_tuple[cnt];
        ejs->T_size[cnt] = tb->tupleNum;
        TupleBuffer::destructor(tb);

        cnt++;
    }
    return NULL;
}

__global__ void nLJ(struct Tuple* d_a, struct Tuple* d_b, long n_a, long n_b, struct Result* res)
{
    // int x = threadIdx.x + blockIdx.x * blockDim.x;
    // int y = threadIdx.y + blockIdx.y * blockDim.y;
    // if (x < n_a && y < n_b) {
    //     if (d_a[x].key == d_b[y].key) {
    //         (*res)->key1 = d_b[x].key;
    //         res->dval1 = d_a[x].dval;
    //         res->key2 = d_a[y].key;
    //         res->dval2 = d_a[y].dval;
    //     } else
    //         res = NULL;
    // }
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x >= n_b)
        return;
    for (int i = 0; i < n_a; i++) {
        if (d_b[x].key == d_a[i].key) {
            (res + i * x)->key1 = d_a[i].key;
            (res + i * x)->dval1 = d_a[i].dval;
            (res + i * x)->key2 = d_b[x].key;
            (res + i * x)->dval2 = d_b[y].dval;
        } else {
            (res + i * x)->key1 = -1;
            (res + i * x)->dval1 = -1;
            (res + i * x)->key2 = -1;
            (res + i * x)->dval2 = -1;
        }
    }
}

void moveResulttoHost(void* arg)
{
    ExternalJoinState* ejs = static_cast<ExternalJoinState*>(arg);
    hipError_t cudaStatus =
        hipMemcpy(ejs->res, ejs->d_res, ejs->T_size[0] * ejs->T_size[1] * sizeof(Result), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        /*call error func*/
    }
    for (long i = 0; i < T_size[0] * T_size[1]; i++) {
        if ((ejs->res + i)->key1 != -1) {
            ejs->prb->put((ejs->res + i)->key1, (ejs->res + i)->dval1, (ejs->res + i)->key2, (ejs->res + i)->dval2);
        }
    }
    /*put result to host, then put it to queue*/
}

void nestLoopJoin(void* arg, struct Tuple* d_a, struct Tuple* d_b, long n_a, long n_b, struct Result* res)
{
    ExternalJoinState* ejs = static_cast<ExternalJoinState*>(arg);
    //    struct Result *res, *d_res;
    ejs->res = (struct Result*)malloc(ejs->T_size[0] * ejs->T_size[1] * sizeof(Result));
    hipError_t cudaStatus = hipMalloc((void**)&ejs->d_res[cnt], ejs->T_size[0] * ejs->T_size[1] * sizeof(Result));

    nlJ<<<32, 1024>>>(d_a, d_b, n_a, n_b, d_res);

    cudaStatus = hipDeviceSynchronize();

    // moveResulttoHost(ejs);
}
